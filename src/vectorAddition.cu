#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

//#include <hip/hip_runtime_api.h>

__global__ void vector_add(const float* A, const float* B, float* C, int N) {

  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N) {
    C[i] = A[i] + B[i] + 0.0f;
  }

}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, const float* B, float* C, int N) {
   int threadsPerBlock = 256;
   int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
   vector_add<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
   printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,threadsPerBlock);
   hipDeviceSynchronize();
}


/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
/**
 * Host main routine
 */
int main(void) {
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  // Print the vector length to be used, and compute its size
  int N = 50000;
  size_t size = N * sizeof(float);
  printf("[Vector addition of %d elements]\n", N);

  // Allocate the host input vector A
  float *h_A = (float *)malloc(size);

  // Allocate the host input vector B
  float *h_B = (float *)malloc(size);

  // Allocate the host output vector C
  float *h_C = (float *)malloc(size);

  // Verify that allocations succeeded
  if (h_A == NULL || h_B == NULL || h_C == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
  }

  // Initialize the host input vectors
  for (int i = 0; i < N; ++i) {
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
  }

  // Allocate the device input vector A
  float *d_A = NULL;
  err = hipMalloc((void **)&d_A, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Allocate the device input vector B
  float *d_B = NULL;
  err = hipMalloc((void **)&d_B, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Allocate the device output vector C
  float *d_C = NULL;
  err = hipMalloc((void **)&d_C, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the host input vectors A and B in host memory to the device input
  // vectors in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector B from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Launch the Vector Add CUDA Kernel
  //int threadsPerBlock = 256;
  //int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  //solve<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
  err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector C from device to host (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Verify that the result vector is correct
  for (int i = 0; i < N; ++i) {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Test PASSED\n");

  // Free device global memory
  err = hipFree(d_A);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_B);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_C);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  printf("Done\n");
  return 0;
}
